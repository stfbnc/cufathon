#include "hip/hip_runtime.h"
#include <stdio.h>
#include "ht_kernel.cuh"


__global__
void MFDFAforHTKernel(const double * __restrict__ y, const double * __restrict__ t, int N,
                      const int * __restrict__ winSizes, int nWins,
                      double * __restrict__ flucVec_mfdfa)
{
    int iw = blockIdx.x * blockDim.x + threadIdx.x;

    if(iw < nWins)
    {
        int currWinSize = winSizes[iw];
        int Ns = N / currWinSize;
        double f = 0.0;

        for(int i = 0; i < Ns; i++)
        {
            double rms = 0.0, rms2 = 0.0;
            int startLim = i * currWinSize;
            double m = 0.0, q = 0.0;

            fit(currWinSize, t + startLim, y + startLim, &m, &q);

            for(int j = 0; j < currWinSize; j++)
            {
                double var = y[startLim + j] - (q + m * t[startLim + j]);
                rms += pow(var, 2.0);
            }

            startLim = i * currWinSize + (N - Ns * currWinSize);
            fit(currWinSize, t + startLim, y + startLim, &m, &q);

            for(int j = 0; j < currWinSize; j++)
            {
                double var = y[startLim + j] - (q + m * t[startLim + j]);
                rms2 += pow(var, 2.0);
            }

            f += log(rms / currWinSize) + log(rms2 / currWinSize);
        }

        flucVec_mfdfa[iw] = exp(f / (4.0 * Ns));
    }
}

__global__
void HTKernel(const double * __restrict__ y, const double * __restrict__ t, int N,
              int scale, int prevScale, int Ns, double * __restrict__ flucVec)
{   
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(i < Ns)
    {   
        double f = 0.0; 
        double m = 0.0, q = 0.0;
            
        fit(scale, t + i, y + i, &m, &q);
            
        for(int j = 0; j < scale; j++)
        {   
            double var = y[i + j] - (q + m * t[i + j]);
            f += pow(var, 2.0);
        }
            
        flucVec[prevScale + i] = sqrt(f / scale);
    }
}

__global__
void finalHTKernel(double * __restrict__ vecht, double Ns,
      int scale, int prevScale,
      double *H0, double *H0_intercept)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < Ns)
    {
        double dscale = static_cast<double>(scale);
        vecht[prevScale + i] = (*H0_intercept + *H0 * log(dscale) - log(vecht[prevScale + i])) / (log(Ns) - log(dscale)) + *H0;
    }
}

void cudaHT(double *y, double *t, int N, int *scales, int nScales, double *ht, int nThreads)
{
    // ht variables
    int *prevScales = new int [nScales];
    int sLen = 0;
    for(int i = 0; i < nScales; i++)
    {
        sLen += (N - scales[i] + 1);
        prevScales[i] = 0;
        for(int j = 0; j < i; j++)
        {
            prevScales[i] += (N - scales[j] + 1);
        }
    }

    double *d_ht;
    hipMalloc(&d_ht, sLen * sizeof(double));

    // mfdfa variables
    int nWins = 20;
    int winSizes[nWins];
    int winStep = round((N / 4 - 10) / static_cast<double>(nWins));
    for(int i = 0; i < (nWins - 1); i++)
    {
        winSizes[i] = 10 + i * winStep;
    }
    winSizes[nWins - 1] = N / 4;

    int *d_winSizes;
    hipMalloc(&d_winSizes, nWins * sizeof(int));
    hipMemcpy(d_winSizes, winSizes, nWins * sizeof(int), hipMemcpyHostToDevice);

    double *flucVec_mfdfa;
    hipMalloc(&flucVec_mfdfa, nWins * sizeof(double));

    // kernel parameters
    dim3 threadsPerBlock_mfdfa(nThreads);
    dim3 blocksPerGrid_mfdfa((nWins + nThreads - 1) / nThreads);
    dim3 threadsPerBlock(nThreads);

    hipStream_t stream_1, stream_2;
    hipStreamCreate(&stream_1);
    hipStreamCreate(&stream_2);

    // kernels
    MFDFAforHTKernel<<<blocksPerGrid_mfdfa, threadsPerBlock_mfdfa, 0, stream_1>>>(y, t, N, d_winSizes, nWins, flucVec_mfdfa);
    for(int i = 0; i < nScales; i++)
    {
        int Ns = N - scales[i] + 1;
        dim3 blocksPerGrid((Ns + nThreads - 1) / nThreads);
        HTKernel<<<blocksPerGrid, threadsPerBlock, 0, stream_2>>>(y, t, N, scales[i], prevScales[i], Ns, d_ht);
    }
    hipDeviceSynchronize();

    // log variables for fit
    double *d_logW_mfdfa, *d_logF_mfdfa;
    hipMalloc(&d_logW_mfdfa, nWins * sizeof(double));
    hipMalloc(&d_logF_mfdfa, nWins * sizeof(double));

    doubleToLog<<<blocksPerGrid_mfdfa, threadsPerBlock_mfdfa, 0, stream_1>>>(flucVec_mfdfa, d_logF_mfdfa, nWins);
    intToLog<<<blocksPerGrid_mfdfa, threadsPerBlock_mfdfa, 0, stream_2>>>(d_winSizes, d_logW_mfdfa, nWins);

    hipStreamDestroy(stream_1);
    hipStreamDestroy(stream_2);

    // mfdfa fit
    double *d_H_mfdfa, *d_I_mfdfa;
    hipMalloc(&d_H_mfdfa, sizeof(double));
    hipMalloc(&d_I_mfdfa, sizeof(double));
    hFit<<<1, 1>>>(nWins, d_logW_mfdfa, d_logF_mfdfa, d_H_mfdfa, d_I_mfdfa);
    hipDeviceSynchronize();

    // ht
    for(int i = 0; i < nScales; i++)
    {
        double Ns = N - scales[i] + 1;
        dim3 blocksPerGrid((Ns + nThreads - 1) / nThreads);
        finalHTKernel<<<blocksPerGrid, threadsPerBlock>>>(d_ht, Ns, scales[i], prevScales[i], d_H_mfdfa, d_I_mfdfa);
    }

    // copy to host
    hipMemcpy(ht, d_ht, sLen * sizeof(double), hipMemcpyDeviceToHost);

    // free memory
    delete [] prevScales;

    hipFree(d_winSizes);
    hipFree(flucVec_mfdfa);
    hipFree(d_logW_mfdfa);
    hipFree(d_logF_mfdfa);
    hipFree(d_H_mfdfa);
    hipFree(d_I_mfdfa);
}

