#include "hip/hip_runtime.h"
#include <stdio.h>
#include "dfa_kernel.cuh"


__global__
void DFAKernel(const double * __restrict__ y, const double * __restrict__ t, int N,
               const int * __restrict__ winSizes, int nWins, double * __restrict__ flucVec)
{
    int nWin = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(nWin < nWins)
    {
        int currWinSize = winSizes[nWin];
        int Ns = N / currWinSize;
        double f = 0.0;
        
        for(int i = 0; i < Ns; i++)
        {
            int startLim = i * currWinSize;
            double m = 0.0, q = 0.0;

            fit(currWinSize, t + startLim, y + startLim, &m, &q);

            for(int j = 0; j < currWinSize; j++)
            {
                double var = y[startLim + j] - (q + m * t[startLim + j]);
                f += pow(var, 2.0);
            }
        }

        flucVec[nWin] = sqrt(f / (Ns * currWinSize));
    }
}

void cudaDFA(double *y, double *t, int N, int *winSizes, int nWins, double *flucVec, double *I, double *H, int nThreads)
{
    hipError_t cudaErr;

    dim3 threadsPerBlock(nThreads);
    dim3 blocksPerGrid((nWins + nThreads - 1) / nThreads);
    DFAKernel<<<blocksPerGrid, threadsPerBlock>>>(y, t, N, winSizes, nWins, flucVec);
    hipDeviceSynchronize();

    double *logW, *logF;
    cudaErr = hipMalloc(&logW, nWins * sizeof(double));
    if(cudaErr != hipSuccess)
        fprintf(stderr, "%s\n", hipGetErrorString(cudaErr));
    cudaErr = hipMalloc(&logF, nWins * sizeof(double));
    if(cudaErr != hipSuccess)
        fprintf(stderr, "%s\n", hipGetErrorString(cudaErr));
    
    hipStream_t stream_1, stream_2;
    hipStreamCreate(&stream_1);
    hipStreamCreate(&stream_2);
    doubleToLog<<<blocksPerGrid, threadsPerBlock, 0, stream_1>>>(flucVec, logF, nWins);
    intToLog<<<blocksPerGrid, threadsPerBlock, 0, stream_2>>>(winSizes, logW, nWins);
    hipStreamDestroy(stream_1);
    hipStreamDestroy(stream_2);

    double *d_H, *d_I;
    cudaErr = hipMalloc(&d_H, sizeof(double));
    if(cudaErr != hipSuccess)
        fprintf(stderr, "%s\n", hipGetErrorString(cudaErr));
    cudaErr = hipMalloc(&d_I, sizeof(double));
    if(cudaErr != hipSuccess)
        fprintf(stderr, "%s\n", hipGetErrorString(cudaErr));
    hFit<<<1, 1>>>(nWins, logW, logF, d_H, d_I);
    hipDeviceSynchronize();

    cudaErr = hipMemcpy(I, d_I, sizeof(double), hipMemcpyDeviceToHost);
    if(cudaErr != hipSuccess)
        fprintf(stderr, "%s\n", hipGetErrorString(cudaErr));
    cudaErr = hipMemcpy(H, d_H, sizeof(double), hipMemcpyDeviceToHost);
    if(cudaErr != hipSuccess)
        fprintf(stderr, "%s\n", hipGetErrorString(cudaErr));

    cudaErr = hipFree(d_H);
    if(cudaErr != hipSuccess)
        fprintf(stderr, "%s\n", hipGetErrorString(cudaErr));
    cudaErr = hipFree(d_I);
    if(cudaErr != hipSuccess)
        fprintf(stderr, "%s\n", hipGetErrorString(cudaErr));

    fprintf(stderr, "I = %lf, H = %lf\n", *I, *H);
}

__global__
void DFAKernelInner(const double * __restrict__ y, const double * __restrict__ t, int N,
                    const int * __restrict__ winSizes, int nWins, double * __restrict__ flucVec)
{
    int nWin = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ double sh[];
    double *s_y = &sh[0];
    double *s_t = &sh[N];

    if(nWin == 0)
    {
        for(int i = 0; i < N; i++)
        {
            s_y[i] = y[i];
            s_t[i] = t[i];
        }
    }

    __syncthreads();

    if(nWin < nWins)
    {   
        int currWinSize = winSizes[nWin];
        int Ns = N / currWinSize;
        double f = 0.0;
        
        for(int i = 0; i < Ns; i++)
        {   
            int startLim = i * currWinSize;
            double m = 0.0, q = 0.0;
            
            fit(currWinSize, s_t + startLim, s_y + startLim, &m, &q);
            
            for(int j = 0; j < currWinSize; j++)
            {   
                double var = s_y[startLim + j] - (q + m * s_t[startLim + j]);
                f += pow(var, 2.0);
            }
        }
        
        flucVec[nWin] = sqrt(f / (Ns * currWinSize));
    }
}

void cudaDFAInner(double *y, double *t, int N, int *winSizes, int nWins, double *flucVec, int nThreads)
{
    dim3 threadsPerBlock(nThreads);
    dim3 blocksPerGrid((nWins + nThreads - 1) / nThreads);
    DFAKernelInner<<<blocksPerGrid, threadsPerBlock, 2 * N * sizeof(double)>>>(y, t, N, winSizes, nWins, flucVec);
    hipDeviceSynchronize();
}
