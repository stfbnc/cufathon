#include "hip/hip_runtime.h"
#include "dfa_kernel.cuh"


__global__
void DFAKernel(const float * __restrict__ y, const float * __restrict__ t, int N,
               const int * __restrict__ winSizes, int nWins, float * __restrict__ flucVec)
{
    int nWin = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(nWin < nWins)
    {
        int currWinSize = winSizes[nWin];
        int Ns = N / currWinSize;
        float f = 0.0;
        
        for(int i = 0; i < Ns; i++)
        {
            int startLim = i * currWinSize;
            float m = 0.0, q = 0.0;

            fit(currWinSize, t + startLim, y + startLim, &m, &q);

            for(int j = 0; j < currWinSize; j++)
            {
                float var = y[startLim + j] - (q + m * t[startLim + j]);
                f += pow(var, 2.0f);
            }
        }

        flucVec[nWin] = sqrt(f / (Ns * currWinSize));
    }
}

__global__
void DFAKernelBackwards(const float * __restrict__ y, const float * __restrict__ t, int N,
                        const int * __restrict__ winSizes, int nWins, float * __restrict__ flucVec)
{
    int nWin = blockIdx.x * blockDim.x + threadIdx.x;

    if(nWin < nWins)
    {
        int currWinSize = winSizes[nWin];
        int Ns = N / currWinSize;
        float f = 0.0;

        for(int i = 0; i < Ns; i++)
        {
            int startLim = i * currWinSize;
            float m = 0.0, q = 0.0;

            fit(currWinSize, t + startLim, y + startLim, &m, &q);

            for(int j = 0; j < currWinSize; j++)
            {
                float var = y[startLim + j] - (q + m * t[startLim + j]);
                f += pow(var, 2.0f);
            }

            startLim = i * currWinSize + (N - Ns * currWinSize);
            fit(currWinSize, t + startLim, y + startLim, &m, &q);

            for(int j = 0; j < currWinSize; j++)
            {
                float var = y[startLim + j] - (q + m * t[startLim + j]);
                f += pow(var, 2.0f);
            }
        }

        flucVec[nWin] = sqrt(f / (2.0f * Ns * currWinSize));
    }
}

void cudaDFA(float *y, float *t, int N, int *winSizes, int nWins, bool revSeg, float *flucVec, float *I, float *H, int nThreads)
{
    // device variables
    float *d_flucVec;
    hipMalloc(&d_flucVec, nWins * sizeof(float));

    int *d_winSizes;
    hipMalloc(&d_winSizes, nWins * sizeof(int));

    // copy to device
    hipMemcpy(d_winSizes, winSizes, nWins * sizeof(int), hipMemcpyHostToDevice);

    // dfa kernel
    dim3 threadsPerBlock(nThreads);
    dim3 blocksPerGrid((nWins + nThreads - 1) / nThreads);
    if(revSeg)
    {
        DFAKernelBackwards<<<blocksPerGrid, threadsPerBlock>>>(y, t, N, d_winSizes, nWins, d_flucVec);
    }
    else
    {
        DFAKernel<<<blocksPerGrid, threadsPerBlock>>>(y, t, N, d_winSizes, nWins, d_flucVec);
    }

    // device variables
    float *d_logW, *d_logF;
    hipMalloc(&d_logW, nWins * sizeof(float));
    hipMalloc(&d_logF, nWins * sizeof(float));
    
    // log transforms
    hipStream_t stream_1, stream_2;
    hipStreamCreate(&stream_1);
    hipStreamCreate(&stream_2);
   
    floatToLog<<<blocksPerGrid, threadsPerBlock, 0, stream_1>>>(d_flucVec, d_logF, nWins);
    intToLog<<<blocksPerGrid, threadsPerBlock, 0, stream_2>>>(d_winSizes, d_logW, nWins);
   
    hipStreamDestroy(stream_1);
    hipStreamDestroy(stream_2);

    // device variables
    float *d_H, *d_I;
    hipMalloc(&d_H, sizeof(float));
    hipMalloc(&d_I, sizeof(float));
   
    // fit kernel
    hFit<<<1, 1>>>(nWins, d_logW, d_logF, d_H, d_I);

    // copy to host
    hipMemcpy(flucVec, d_flucVec, nWins * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(I, d_I, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(H, d_H, sizeof(float), hipMemcpyDeviceToHost);

    // free memory
    hipFree(d_flucVec);
    hipFree(d_winSizes);
    hipFree(d_logW);
    hipFree(d_logF);
    hipFree(d_H);
    hipFree(d_I);
}

__global__
void DFAKernelInner(const float * __restrict__ y, const float * __restrict__ t, int N,
                    const int * __restrict__ winSizes, int nWins, float * __restrict__ flucVec)
{
    int nWin = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ float sh[];
    float *s_y = &sh[0];
    float *s_t = &sh[N];

    if(nWin == 0)
    {
        for(int i = 0; i < N; i++)
        {
            s_y[i] = y[i];
            s_t[i] = t[i];
        }
    }

    __syncthreads();

    if(nWin < nWins)
    {   
        int currWinSize = winSizes[nWin];
        int Ns = N / currWinSize;
        float f = 0.0;
        
        for(int i = 0; i < Ns; i++)
        {   
            int startLim = i * currWinSize;
            float m = 0.0, q = 0.0;
            
            fit(currWinSize, s_t + startLim, s_y + startLim, &m, &q);
            
            for(int j = 0; j < currWinSize; j++)
            {   
                float var = s_y[startLim + j] - (q + m * s_t[startLim + j]);
                f += pow(var, 2.0f);
            }
        }
        
        flucVec[nWin] = sqrt(f / (Ns * currWinSize));
    }
}

