#include "hip/hip_runtime.h"
#include <stdio.h>
#include "utils_kernels.cuh"


__global__
void linRangeKernel(float * __restrict__ vec, int N, int start)
{
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(tx < N)
    {
        vec[tx] = tx + start;
    }
}

void linRange(float *vec, int N, int start)
{
    int nThreads = 512;
    dim3 threadsPerBlock(nThreads);
    dim3 blocksPerGrid((N + nThreads - 1) / nThreads);
    linRangeKernel<<<blocksPerGrid, threadsPerBlock>>>(vec, N, start);
    hipDeviceSynchronize();
}

__global__
void floatToLog(const float * __restrict__ vec, float * __restrict__ logVec, int N)
{
    int tx = blockIdx.x * blockDim.x + threadIdx.x;

    if(tx < N)
    {
        logVec[tx] = log(vec[tx]);
    } 
}

__global__
void intToLog(const int * __restrict__ vec, float * __restrict__ logVec, int N)
{
    int tx = blockIdx.x * blockDim.x + threadIdx.x;

    if(tx < N)
    {
        logVec[tx] = log(1.0f * vec[tx]);
    }
}

