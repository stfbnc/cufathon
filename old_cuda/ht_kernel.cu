#include "hip/hip_runtime.h"
#include <stdio.h>
#include "ht_kernel.cuh"


__global__
void MFDFAforHTKernel(const float * __restrict__ y, const float * __restrict__ t, int N,
                      const int * __restrict__ winSizes, int nWins,
                      float * __restrict__ flucVec_mfdfa)
{
    int iw = blockIdx.x * blockDim.x + threadIdx.x;

    if(iw < nWins)
    {
        int currWinSize = winSizes[iw];
        int Ns = N / currWinSize;
        float f = 0.0;

        for(int i = 0; i < Ns; i++)
        {
            float rms = 0.0, rms2 = 0.0;
            int startLim = i * currWinSize;
            float m = 0.0, q = 0.0;

            fit(currWinSize, t + startLim, y + startLim, &m, &q);

            for(int j = 0; j < currWinSize; j++)
            {
                float var = y[startLim + j] - (q + m * t[startLim + j]);
                rms += pow(var, 2.0f);
            }

            startLim = i * currWinSize + (N - Ns * currWinSize);
            fit(currWinSize, t + startLim, y + startLim, &m, &q);

            for(int j = 0; j < currWinSize; j++)
            {
                float var = y[startLim + j] - (q + m * t[startLim + j]);
                rms2 += pow(var, 2.0f);
            }

            f += log(rms / currWinSize) + log(rms2 / currWinSize);
        }

        flucVec_mfdfa[iw] = exp(f / (4.0f * Ns));
    }
}

__global__
void HTKernel(const float * __restrict__ y, const float * __restrict__ t, int N,
              int scale, int prevScale, int Ns, float * __restrict__ flucVec)
{   
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(i < Ns)
    {   
        float f = 0.0; 
        float m = 0.0, q = 0.0;
            
        fit(scale, t + i, y + i, &m, &q);
            
        for(int j = 0; j < scale; j++)
        {   
            float var = y[i + j] - (q + m * t[i + j]);
            f += pow(var, 2.0f);
        }
            
        flucVec[prevScale + i] = sqrt(f / scale);
    }
}

__global__
void finalHTKernel(float * __restrict__ vecht, float Ns,
      int scale, int prevScale,
      float *H0, float *H0_intercept)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < Ns)
    {
        float dscale = static_cast<float>(scale);
        vecht[prevScale + i] = (*H0_intercept + *H0 * log(dscale) - log(vecht[prevScale + i])) / (log(Ns) - log(dscale)) + *H0;
    }
}

void cudaHT(float *y, float *t, int N, int *scales, int nScales, float *ht, int nThreads)
{
    // ht variables
    int *prevScales = new int [nScales];
    int sLen = 0;
    for(int i = 0; i < nScales; i++)
    {
        sLen += (N - scales[i] + 1);
        prevScales[i] = 0;
        for(int j = 0; j < i; j++)
        {
            prevScales[i] += (N - scales[j] + 1);
        }
    }

    float *d_ht;
    hipMalloc(&d_ht, sLen * sizeof(float));

    // mfdfa variables
    int nWins = 20;
    int winSizes[nWins];
    int winStep = round((N / 4 - 10) / static_cast<float>(nWins));
    for(int i = 0; i < (nWins - 1); i++)
    {
        winSizes[i] = 10 + i * winStep;
    }
    winSizes[nWins - 1] = N / 4;

    int *d_winSizes;
    hipMalloc(&d_winSizes, nWins * sizeof(int));
    hipMemcpy(d_winSizes, winSizes, nWins * sizeof(int), hipMemcpyHostToDevice);

    float *flucVec_mfdfa;
    hipMalloc(&flucVec_mfdfa, nWins * sizeof(float));

    // kernel parameters
    dim3 threadsPerBlock_mfdfa(nWins); //nThreads);
    dim3 blocksPerGrid_mfdfa(1); //(nWins + nThreads - 1) / nThreads);
    dim3 threadsPerBlock(nThreads);

    hipStream_t stream_1, stream_2;
    hipStreamCreate(&stream_1);
    hipStreamCreate(&stream_2);

    // kernels
    MFDFAforHTKernel<<<blocksPerGrid_mfdfa, threadsPerBlock_mfdfa, 0, stream_1>>>(y, t, N, d_winSizes, nWins, flucVec_mfdfa);
    for(int i = 0; i < nScales; i++)
    {
        int Ns = N - scales[i] + 1;
        dim3 blocksPerGrid((Ns + nThreads - 1) / nThreads);
        HTKernel<<<blocksPerGrid, threadsPerBlock, 0, stream_2>>>(y, t, N, scales[i], prevScales[i], Ns, d_ht);
    }
    hipDeviceSynchronize();

    // log variables for fit
    float *d_logW_mfdfa, *d_logF_mfdfa;
    hipMalloc(&d_logW_mfdfa, nWins * sizeof(float));
    hipMalloc(&d_logF_mfdfa, nWins * sizeof(float));

    floatToLog<<<blocksPerGrid_mfdfa, threadsPerBlock_mfdfa, 0, stream_1>>>(flucVec_mfdfa, d_logF_mfdfa, nWins);
    intToLog<<<blocksPerGrid_mfdfa, threadsPerBlock_mfdfa, 0, stream_2>>>(d_winSizes, d_logW_mfdfa, nWins);

    hipStreamDestroy(stream_1);
    hipStreamDestroy(stream_2);

    // mfdfa fit
    float *d_H_mfdfa, *d_I_mfdfa;
    hipMalloc(&d_H_mfdfa, sizeof(float));
    hipMalloc(&d_I_mfdfa, sizeof(float));
    hFit<<<1, 1>>>(nWins, d_logW_mfdfa, d_logF_mfdfa, d_H_mfdfa, d_I_mfdfa);
    hipDeviceSynchronize();

    // ht
    for(int i = 0; i < nScales; i++)
    {
        float Ns = N - scales[i] + 1;
        dim3 blocksPerGrid((Ns + nThreads - 1) / nThreads);
        finalHTKernel<<<blocksPerGrid, threadsPerBlock>>>(d_ht, Ns, scales[i], prevScales[i], d_H_mfdfa, d_I_mfdfa);
    }

    // copy to host
    hipMemcpy(ht, d_ht, sLen * sizeof(float), hipMemcpyDeviceToHost);

    // free memory
    delete [] prevScales;

    hipFree(d_winSizes);
    hipFree(flucVec_mfdfa);
    hipFree(d_logW_mfdfa);
    hipFree(d_logF_mfdfa);
    hipFree(d_H_mfdfa);
    hipFree(d_I_mfdfa);
}

